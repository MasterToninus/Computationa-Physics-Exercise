/*//------------------------------------------------------------------------------------------------------------
| TEST delle funzioni di base della classe (funzionalita'):
|	inizializzazione della matrice
|       print
|	inizializzazione Random
|	resize
*///------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int n1=5;
	int n2=5;
	//cout<<" #righe = ";cin>>n1;cout<<endl;
	//cout<<" #colonne = ";cin>>n2;cout<<endl;




	//Eventi per le statistiche
	hipEvent_t T1, T2, T3, T4;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	hipEventCreate(&T3);
	hipEventCreate(&T4);
	float diff_time;

	matrice matA (n1,n2);

	cout<<"matrice iniziale con GPU"<<endl;
	hipEventRecord(T3,0);
	matA.Gpu_Init();
	matA.sync_DeviceToHost();

	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	cout << "tempo=" << diff_time<<"\n";
	matA.Gpu_print();

	cout<<"matrice iniziale con CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Init();
	matA.sync_HostToDevice();
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	matA.Cpu_print();

	cout<<"matrice Hilbert con GPU"<<endl;
	hipEventRecord(T3,0);
	matA.Gpu_Hilbert_Init();
	matA.sync_DeviceToHost();

	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	cout << "tempo=" << diff_time<<"\n";
	matA.Gpu_print();

	cout<<"matrice Hilbert con CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Hilbert_Init();
	matA.sync_HostToDevice();
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	matA.Cpu_print();



	cout<<"matrice iniziale random con GPU"<<endl;
	hipEventRecord(T3,0);
	matA.Gpu_Init_Rand(0.95,1.05);
	matA.sync_DeviceToHost();

	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	cout << "tempo=" << diff_time<<"\n";
	matA.Gpu_print();

	cout<<"matrice iniziale random con CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Init_Rand(0.95,1.05);
	matA.sync_HostToDevice();

	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	matA.Cpu_print();

	cout<<"Resize e Rinit con GPU"<<endl;
	hipEventRecord(T3,0);
	matA.Resize(10,10);
	matA.Gpu_Init_Rand(0.95,1.05);
	matA.sync_DeviceToHost();

	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	cout << "tempo=" << diff_time<<"\n";
	matA.Gpu_print();

	cout<<"Resize con CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Resize(10,10);
	matA.Cpu_Init_Rand(0.95,1.05);
	matA.sync_HostToDevice();

	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	matA.Cpu_print();

cout<<"Commenti:\nDa questa implementazione banale dell'inizializzazione si scopre che il hiprand della gpu non e' particolarmente efficiente, il motivo e' imputabile alla non ideale ottimizzazione del numero di thread scelto per la chiamata del kernel."<<endl;


	hipEventDestroy(T1);
	hipEventDestroy(T2);
	hipEventDestroy(T3);
	hipEventDestroy(T4);

  return 0;
}

