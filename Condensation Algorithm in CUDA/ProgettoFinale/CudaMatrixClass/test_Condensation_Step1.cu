/*//------------------------------------------------------------------------------------------------------------
| TEST tempistiche del metodo di Step di condensazione per matrici pivotizzate, ovvero con elemento angolo basso destra =1
|
*///------------------------------------------------------------------------------------------------------------



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int device_id =0;
	hipSetDevice(device_id);
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, device_id);
	printf("Scheda Utilizzata:  %s \n\n", prop.name);

	int n=5000;

	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;


	matrice matA (n);
	matA.Init();
	matA[n-1][n-1]=1;
	matA.sync_HostToDevice();
	//matA.print();


	cout<<"\n Condensation step secondo la cpu "<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Step_Condensation_Simple();
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	//matA.Cpu_print();
	cout << "tempo=" << diff_time<<"\n";

	cout<<"\n Condensation step secondo la Gpu"<<endl;
	matrice matB (n);
	matB.Init();
	matB[n-1][n-1]=1;
	matB.sync_HostToDevice();
	//matB.print();
	hipEventRecord(T1,0);
	matB.Gpu_Step_Condensation_Simple();
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	//matB.Gpu_print();
	cout << "tempo=" << diff_time<<"\n";




	cout<<"\n Condensation step secondo la Gpu (texture ver.)"<<endl;
	matrice matC (n);
	matC.Init();
	matC[n-1][n-1]=1;
	matC.sync_HostToDevice();
	//matC.print();
	hipEventRecord(T1,0);
	matC.Gpu_Step_Condensation_Simple_Best();
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	//matC.Gpu_print();
	cout << "tempo=" << diff_time<<"\n";



	hipEventDestroy(T1);
	hipEventDestroy(T2);

  return 0;
}

