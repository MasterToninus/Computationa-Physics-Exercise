/*//------------------------------------------------------------------------------------------------------------
| TEST funzionamento del metodo di Condensazione Completo con la Matrice di Hilbert, nota per mettere alla prova la stabilita' numerica
| al crescere di N la matrice ha determinante molto piccolo ma diverso da 0!
*///------------------------------------------------------------------------------------------------------------



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int n=10;

	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;


	matrice matA (n);
	matA.Hilbert_Init();
	matA.print();
	matA.print_tomath("Output/file.dat");

	float pivot[n];
	float result;

	cout<<"\n Condensation Procedura Secondo la CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Condensation(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	for(int i=0; i<n;i++)cout<<pivot[i]<<endl;
	cout << "tempo=" << diff_time<<"\n";

//	matA.print();
	result=matA.Cpu_Determinant_Condensation();
	cout<<"\n Determinante ="<<result<<endl;

	matA.sync_DeviceToHost();

	cout<<"\n Condensation Procedura Secondo la GPU "<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Condensation(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	for(int i=0; i<n;i++)cout<<pivot[i]<<endl;
	cout << "tempo=" << diff_time<<"\n";
//	matA.print();
	result=matA.Gpu_Determinant_Condensation();
	cout<<"\n Determinante ="<<result<<endl;

	matA.sync_HostToDevice();

	cout<<"\n Condensation Procedura Secondo la GPU  (versione TEXTURE)"<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Condensation_Best(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	for(int i=0; i<n;i++)cout<<pivot[i]<<endl;
	cout << "tempo=" << diff_time<<"\n";
//	matA.print();
	result=matA.Gpu_Determinant_Condensation_Best();
	cout<<"\n Determinante ="<<result<<endl;




  return 0;
}

