/*//------------------------------------------------------------------------------------------------------------
| TEST funzionamento del metodo di pivoting (trova massimo e poi divide)
|
*///------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int n1=3;
	int n2=4;
	//cout<<" #righe = ";cin>>n1;cout<<endl;
	//cout<<" #colonne = ";cin>>n2;cout<<endl;

	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;

	matrice matA (n1,n2);
	matA.Init_Rand(-5.0,5.0);
	cout<<"Matrice : \n";
	matA.print();

	int pivot_idx;
	hipEventRecord(T1,0);
	matA.Cpu_Max_Pivoting(&pivot_idx);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout<<"Pivot secondo la cpu: A( "<<n1-1<<" , "<<pivot_idx<<" ) = "<<matA.Cpu_pick(n1-1,pivot_idx)<<endl;
	cout << "tempo=" << diff_time<<"\n";

	hipEventRecord(T1,0);
	matA.Gpu_Max_Pivoting(&pivot_idx);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout<<"Pivot secondo la Gpu: A( "<<n1-1<<" , "<<pivot_idx<<" ) = "<<matA.Gpu_pick(n1-1,pivot_idx)<<endl;
	cout << "tempo=" << diff_time<<"\n";

	float pivot_value;
	hipEventRecord(T1,0);
	matA.Gpu_Max_Pivoting_Speedy(&pivot_idx, &pivot_value);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout<<"Pivot secondo la Gpu speedy: A( "<<n1-1<<" , "<<pivot_idx<<" ) = "<<pivot_value<<endl;
	cout << "tempo=" << diff_time<<"\n";

	cout<<"Divido per il Pivot secondo la Gpu"<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Division(pivot_idx);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	matA.Gpu_print();

	cout<<"Divido per il Pivot secondo la Cpu"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Division(pivot_idx);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	matA.Cpu_print();



  return 0;
}

