/*//------------------------------------------------------------------------------------------------------------
| TEST funzionamento del metodo di Condensazione Completo
|
*///------------------------------------------------------------------------------------------------------------



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int n=10;

	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;


	matrice matA (n);
	matA.Init_Rand(-10,10);
	matA[n-1][n-1]=1;
	matA[0][1]=0;
	matA.sync_HostToDevice();
	matA.print();
	matA.print_tomath("Output/file.dat");

	float pivot[n];
	float result;

	cout<<"\n Condensation Procedura Secondo la CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Condensation(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
	for(int i=0; i<n;i++)cout<<pivot[i]<<endl;

//	matA.print();
	result=matA.Cpu_Determinant_Condensation();
	cout<<"\n Determinante ="<<result<<endl;

	matA.sync_DeviceToHost();

	cout<<"\n Condensation Procedura Secondo la GPU "<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Condensation(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
//	matA.print();
	result=matA.Gpu_Determinant_Condensation();
	cout<<"\n Determinante ="<<result<<endl;
	for(int i=0; i<n;i++)cout<<pivot[i]<<endl;

	matA.sync_HostToDevice();

	cout<<"\n Condensation Procedura Secondo la GPU  (versione TEXTURE)"<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Condensation_Best(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
//	matA.print();
	result=matA.Gpu_Determinant_Condensation_Best();
	cout<<"\n Determinante ="<<result<<endl;
	for(int i=0; i<n;i++)cout<<pivot[i]<<endl;




  return 0;
}

