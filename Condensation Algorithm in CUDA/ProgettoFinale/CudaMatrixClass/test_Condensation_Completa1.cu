/*//------------------------------------------------------------------------------------------------------------
| TEST Tempistica del metodo di Condensazione Completo
|
*///------------------------------------------------------------------------------------------------------------



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int device_id =0;
	hipSetDevice(device_id);
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, device_id);
	printf("Scheda Utilizzata:  %s \n\n", prop.name);

	int n=3000;

	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;


	matrice matA (n);
	matA.Init_Rand(-10,10);
	matA[n-1][n-1]=1;
	matA[0][1]=0;
	matA.sync_HostToDevice();
	//matA.print();
	//matA.print_tomath("Output/file.dat");

	float pivot[n];
	//float result;

	cout<<"\n Condensation Procedura Secondo la CPU"<<endl;
	hipEventRecord(T1,0);
	matA.Cpu_Condensation(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";

//	matA.print();
	//result=matA.Cpu_Determinant_Condensation();
	//cout<<"\n Determinante ="<<result<<endl;

	matA.sync_DeviceToHost();

	cout<<"\n Condensation Procedura Secondo la GPU "<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Condensation(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
//	matA.print();
	//result=matA.Ibrid_Determinant_Condensation();
	//cout<<"\n Determinante ="<<result<<endl;

	matA.sync_HostToDevice();

	cout<<"\n Condensation Procedura Secondo la GPU (versione TEXTURE)"<<endl;
	hipEventRecord(T1,0);
	matA.Gpu_Condensation_Best(pivot);
	hipEventRecord(T2,0);
  	hipEventSynchronize(T2);
	hipEventElapsedTime(&diff_time,T1,T2);
	cout << "tempo=" << diff_time<<"\n";
//	matA.print();
	//result=matA.Ibrid_Determinant_Condensation_Best();
	//cout<<"\n Determinante ="<<result<<endl;




  return 0;
}

