/*//------------------------------------------------------------------------------------------------------------
| TEST funzionamento del metodo di swap necessario per il pivoting
|
*///------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int n1=3;
	int n2=4;
	//cout<<" #righe = ";cin>>n1;cout<<endl;
	//cout<<" #colonne = ";cin>>n2;cout<<endl;

	//Eventi per le statistiche
	hipEvent_t T1, T2, T3, T4;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	hipEventCreate(&T3);
	hipEventCreate(&T4);
	float diff_time;


	matrice matA (n1,n2);
	matA.Init_Rand(-5.0,5.0);
	matA.print();


	cout<<"\n SWAP RIGA 0 1, secondo la cpu"<<endl;
	hipEventRecord(T3,0);
	matA.Cpu_Swap_Row(0,1);
	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	matA.Cpu_print();
	cout << "tempo=" << diff_time<<"\n";

	cout<<"\n SWAP Col 0 1, secondo la cpu"<<endl;
	hipEventRecord(T3,0);
	matA.Cpu_Swap_Col(0,1);
	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	matA.Cpu_print();
	cout << "tempo=" << diff_time<<"\n";

	cout<<"\n SWAP RIGA 0 1, secondo la gpu"<<endl;
	hipEventRecord(T3,0);
	matA.Gpu_Swap_Row(0,1);
	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	matA.Gpu_print();
	cout << "tempo=" << diff_time<<"\n";

	cout<<"\n SWAP Col 0 1, secondo la gpu"<<endl;
	hipEventRecord(T3,0);
	matA.Gpu_Swap_Col(0,1);
	hipEventRecord(T4,0);
  	hipEventSynchronize(T4);
	hipEventElapsedTime(&diff_time,T3,T4);
	matA.Gpu_print();
	cout << "tempo=" << diff_time<<"\n";


  return 0;
}

