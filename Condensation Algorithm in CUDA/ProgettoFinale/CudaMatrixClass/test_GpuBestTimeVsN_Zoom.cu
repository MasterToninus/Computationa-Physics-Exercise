/*//------------------------------------------------------------------------------------------------------------
| TEST del tempo d'esecuzione della routine Gpu Texture in funzione del numero N di elementi per lato
|	
|       
*///------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Ausiliary/Statistics.h"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	int device_id =0;
	hipSetDevice(device_id);
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, device_id);
	printf("Scheda Utilizzata:  %s \n\n", prop.name);

	//Parametri per il grafico
	int Nmax =600;
	int Nstep =10;
	ofstream myfile;
  	myfile.open ("Output/GpuBestTimeVsN_Zoom.dat",ofstream::app);
	int n=10;
	float Dati[n];

	float media;
	float errore;


	//Eventi per le statistiche
	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;



	 matrice matA (1,1);


	for(int i=50; i<Nmax; i+=Nstep){
		float pivot[i];
		for(int k=0; k<n; k++){
			matA.Resize(i,i);
 			matA.Cpu_Init_Rand(-10,10);
			matA.sync_HostToDevice();

			hipEventRecord(T1,0);	
			matA.Gpu_Condensation_Best(pivot);
 			hipEventRecord(T2,0);
			hipEventSynchronize(T2);
      			hipEventElapsedTime(&diff_time,T1,T2);
        		//cout << "tempo=" << diff_time<<"\n";
			Dati[k]=diff_time;
		}
		media = Media(Dati,n);
		errore = Errore(Dati,n, media);
		cout<< " N = "<<i<<" \t time = "<<media<< " +o- "<<errore<<endl;
		myfile <<i<<"\t"<<media<<"\t"<<errore<<endl;
	}






	myfile.close();
	hipEventDestroy(T1);
	hipEventDestroy(T2);

  return 0;
}

