/*//------------------------------------------------------------------------------------------------------------
| TEST del tempo d'esecuzione della routine cpu in funzione del numero N di elementi per lato
|	
|       
*///------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


#include "Src/Ausiliary/CudaCrono.cuh"
#include "Src/Ausiliary/Statistics.h"
#include "Src/Cuda_FloatMatrixClass.cuh"

int main(void){

	//Parametri per il grafico
	int Nmax =6100;
	int Nstep =100;
	ofstream myfile;
  	myfile.open ("Output/CpuTimeVsN.dat",ofstream::app);
	int n=10;
	float Dati[n];

	float media;
	float errore;


	//Eventi per le statistiche
	hipEvent_t T1, T2;
        hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;



	 matrice matA (1,1);


	for(int i=100; i<Nmax; i+=Nstep){
		float pivot[i];
		for(int k=0; k<n; k++){
			matA.Resize(i,i);
 			matA.Cpu_Init_Rand(-10,10);


			hipEventRecord(T1,0);	
			matA.Cpu_Condensation(pivot);
 			hipEventRecord(T2,0);
			hipEventSynchronize(T2);
      			hipEventElapsedTime(&diff_time,T1,T2);
        		//cout << "tempo=" << diff_time<<"\n";
			Dati[k]=diff_time;
		}
		media = Media(Dati,n);
		errore = Errore(Dati,n, media);
		cout<< " N = "<<i<<" \t time = "<<media<< " +o- "<<errore<<endl;
		myfile <<i<<"\t"<<media<<"\t"<<errore<<endl;
	}






	myfile.close();
	hipEventDestroy(T1);
	hipEventDestroy(T2);

  return 0;
}

