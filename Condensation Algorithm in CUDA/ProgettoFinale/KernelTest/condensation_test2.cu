#include "hip/hip_runtime.h"
/*//---------------------------------------------------------------------------------------------------------
|
|  Test del miglior Kernel che non utilizza la memoria texture di tutta la matrice ( mostrare l'aumento delle prestazioni con una corretta dimensionalizzazione dei thread)
|
*///---------------------------------------------------------------------------------------------------------
//OGNI thread alloca 5 unsigned int e 2 float ---> 7*4= 28 byte



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


// Numero di thread prudente
#define DIM_GRID 128
#define DIM_BLOCK 128
#define THREAD_PER_BLOCK 128
#include "condensation_tentative.cuh"


/*     DIMENSIONALIZZAZIONE DEI THREAD    */
// Numero di thread "Limite"
#define imin(a,b) (a<b?a:b)

//trucco per ottimizzare il numero di blocchi
//const int blocksPerGrid = imin(32,(N+threadsPerBlock-1)/threadsPerBlock);

#define MAX_DIM_BLOCK 512
#define MAX_DIM_GRID 524288 //in realtà il max numero di thrad è un po' di piu', questa è la potenza di 2 più vicina
#define DIM_WARP 32






//kernel che copia una colonna della matrice in un segmento lineare di memoria
/*
input: 
	puntatore destinatatio
	puntatore matricepartenza
	indice della colonna da copiare
	numero di elementi
*/

__global__ void copy_Col( float *a, float *A,int j1, int numAColumns, int numARow){

  int tid = blockIdx.x*blockDim.x + threadIdx.x; //identità del thread (al solito metto in fila ogni thread a partire dal primo blocco)
  int Nthreads = blockDim.x * gridDim.x;	 //Numero totale di threads istanziati dalla chiamata del kernel

  while(tid<numARow){
	a[tid]=A[tid*numAColumns + j1]; //qui copia
	tid+= Nthreads;	      // se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
	}
  
}

void print(float *A, int ROW, int COL){
		for(int i=0 ; i<ROW ; i++){
			for(int j=0 ; j<COL ; j++){
				//printf(" %.1f ",A[i*COL+j]);
				cout<< A[i*COL+j] << " ";
			}
			cout<<endl;
		}
		cout<<endl;  
}







int main(void){

	int ROW=10000;
	int COL=ROW;
	int N = COL*ROW;
	int Nb= (ROW-1)*(COL-1);
	srand (time(NULL));  

   //Punto La memoria
	float * A_host; float *B_host;
	float * A_dev; float *B_dev;

   //Alloco la matrice
	A_host = new float[N] ();
	B_host = new float[Nb] ();


   //Alloco le matrice device (riservo la memoria richiesta)
	unsigned int ByteSize = N*sizeof(float);
	hipMalloc( (void **)&A_dev, ByteSize  );
	ByteSize = Nb*sizeof(float);
	hipMalloc( (void **)&B_dev, ByteSize  );


   //Riempio la Matrice Host
	//float min = -10, max = 10 , delta = max-min; 
	for(int i=0 ; i<ROW ; i++)for(int j=0 ; j<COL ; j++)A_host[i*COL+j] =  (rand() % 10 );
        A_host[N-1]=1;

   //Riempio La Matrice Device
	hipMemcpy( A_dev, A_host, N*sizeof(float), hipMemcpyHostToDevice);

   //Check Matrice Creata
	//	print(A_host,ROW,COL);



	//Creo degli eventi per la statistica
	hipEvent_t T1,T2;
	hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;



/*
cout<<"CONDENSAZIONE VERSIONE 1"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

   //Kernel
	stepCondensationSimple_v1<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;

cout<<"CONDENSAZIONE VERSIONE 2"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

   //Kernel
	stepCondensationSimple_v2<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;

cout<<"CONDENSAZIONE VERSIONE 3"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);
   //Kernel
	stepCondensationSimple_v3<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1); 
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;
*/
cout<<"CONDENSAZIONE VERSIONE 4 (1 texture)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	float *pivot_Column = 0;
	unsigned int numBytes = ROW*sizeof(float);
	//Alloco la memoria lineare sulla gpu dove caricare la colonna di pivot
	hipMalloc((void**)&pivot_Column, numBytes);

	//Chiamo kernel che mette la colonna in memoria lineare
	copy_Col<<<DIM_GRID,DIM_BLOCK>>>(pivot_Column, A_dev ,COL-1, COL, ROW);

	//Bindo a texture la memoria lineare
	hipBindTexture(NULL, texRef, pivot_Column);
// --------- 


   //Kernel
	stepCondensationSimple_v4<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef );
	hipFree (pivot_Column);
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;


cout<<"CONDENSAZIONE VERSIONE 4 (1 texture), versione ottimizzata"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
//	*pivot_Column = 0;
	numBytes = ROW*sizeof(float);
	//Alloco la memoria lineare sulla gpu dove caricare la colonna di pivot
	hipMalloc((void**)&pivot_Column, numBytes);

	//Chiamo kernel che mette la colonna in memoria lineare
	copy_Col<<<DIM_GRID,DIM_BLOCK>>>(pivot_Column, A_dev ,COL-1, COL, ROW);

	//Bindo a texture la memoria lineare
	hipBindTexture(NULL, texRef, pivot_Column);
// --------- 
	dim3 block(THREAD_PER_BLOCK );  // 512 x 1 x 1	

	unsigned int block_per_row = ROW/THREAD_PER_BLOCK + 1;
	unsigned int block_per_col =  imin(COL, MAX_DIM_GRID / block_per_row);
	dim3 grid( block_per_row, block_per_col );         // 512 x 1 x 1

	cout<<"blocchi : "<<MAX_DIM_BLOCK<<" x 1 x 1 "<<endl;
	cout<<"griglia : "<< block_per_row <<" x "<< block_per_col << " x 1"<<endl;

   //Kernel
	stepCondensationSimple_v4<<<grid,block>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef );
	hipFree (pivot_Column);
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;


cout<<"CONDENSAZIONE VERSIONE 4 (1 texture), versione ottimizzata 2"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
//	*pivot_Column = 0;
	numBytes = ROW*sizeof(float);
	//Alloco la memoria lineare sulla gpu dove caricare la colonna di pivot
	hipMalloc((void**)&pivot_Column, numBytes);

	//Chiamo kernel che mette la colonna in memoria lineare
	copy_Col<<<DIM_GRID,DIM_BLOCK>>>(pivot_Column, A_dev ,COL-1, COL, ROW);

	//Bindo a texture la memoria lineare
	hipBindTexture(NULL, texRef, pivot_Column);
// --------- 
	dim3 block1(THREAD_PER_BLOCK );  // 512 x 1 x 1	

	 block_per_row = ROW/THREAD_PER_BLOCK + 1;
	 block_per_col =1;//  imin(COL, MAX_DIM_GRID / block_per_row);
	dim3 grid1( block_per_row, block_per_col );         // 512 x 1 x 1

	cout<<"blocchi : "<<MAX_DIM_BLOCK<<" x 1 x 1 "<<endl;
	cout<<"griglia : "<< block_per_row <<" x "<< block_per_col << " x 1"<<endl;

   //Kernel
	stepCondensationSimple_v4<<<grid1,block1>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef );
	hipFree (pivot_Column);
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;

cout<<"Conclusione: questo Kernel non trae grande vantaggio dall'utilizzare griglia 2d di blocchi 1d, anzi si ha un grande peggioramento dovuto al fatto che i blocchi incolonnati ricopiano più volte nella shared segmenti di meoria che sarebbe bastato copiare una sola volta in una shared di un blocco che si sviluppa anche in verticale!"<<endl;
/*
cout<<"CONDENSAZIONE VERSIONE 5 (2 texture)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	// *pivot_Column = 0;
	 numBytes = ROW*sizeof(float);
	//Alloco la memoria lineare sulla gpu dove caricare la colonna di pivot
	hipMalloc((void**)&pivot_Column, numBytes);

	//Chiamo kernel che mette la colonna in memoria lineare
	copy_Col<<<DIM_GRID,DIM_BLOCK>>>(pivot_Column, A_dev ,COL-1, COL, ROW);

	//Bindo a texture la memoria lineare
	hipBindTexture(NULL, texRef, pivot_Column);
// --------- 
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------

   //Kernel
	stepCondensationSimple_v5<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);



// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef );
	hipUnbindTexture ( texRef2 );
	hipFree (pivot_Column);
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;




cout<<"CONDENSAZIONE VERSIONE 6 (1 sola  texture grande)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------

   //Kernel
	stepCondensationSimple_v6<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);


// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef2 );
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;
*/


   hipFree( A_dev);
   hipFree( B_dev);



  return 0;
}

