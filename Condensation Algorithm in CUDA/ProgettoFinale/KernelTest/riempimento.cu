/*//---------------------------------------------------------------------------------------------------------
| faccio eseguire un semplice riempimento di vettore alla scheda, riempie con l'indice del thread
| 	
| 
*///---------------------------------------------------------------------------------------------------------






#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>

#define DIM_BLOCK 256
#define DIM_GRID 1024

__global__ void riempimento(float*V, int N){

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int Nthreads = blockDim.x * gridDim.x;	 //Numero totale di threads istanziati dalla chiamata del kernel

  while(tid<N){
	V[tid]=(float)tid;
	tid+= Nthreads;	      // se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
  } 

}




int main(){

//Definisco i puntatori alle variabli host
	float *A_host; 
//Definisco i puntatori alle variabli device
	float *A_dev;

//alloco le variabili host
  	int N=100000000;  
	A_host = new float[N] ();

//Alloco le variabili device
  hipMalloc( (void **)&A_dev, N*sizeof(float)  );

//Setup della configurazione di esecuzione
/*	cudaDeviceProp prop; //cudaDeviceProp e' una struttura di dati, prop e' il nome della variabile di questo tipo
	cudaGetDeviceProperties( &prop, 0); //salva in prop le caratteristiche della prima scheda
	unsigned int Block_Width=prop.maxThreadsDim[0];
	unsigned int Grid_Width=prop.maxGridSize[0];

	dim3 dimBlock(Block_Width,1);
	dim3 dimGrid(Grid_Width,1);
*/

//Riempio le variabili device
	riempimento<<<DIM_GRID,DIM_BLOCK>>>(A_dev,N);
	
//riempio le variabili host
	hipMemcpy( A_host, A_dev, N*sizeof(float), hipMemcpyDeviceToHost);

//controllo!
	for(int i=N-1000; i<N; i++)printf(" %f \n",A_host[i]);


  return 0;
}


