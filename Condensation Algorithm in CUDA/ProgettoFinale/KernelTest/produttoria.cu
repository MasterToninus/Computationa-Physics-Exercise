#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "lock.h"


//va compilato cosi'; nvcc -arch=sm_20 -I/home/vicini/NVIDIA_GPU_Computing_SDK/CUDALibraries/common/inc/ -I/home/vicini/NVIDIA_GPU_Computing_SDK/shared/inc/ -I../../common/ -I/usr/local/cuda/include/ produttoria.cu

const int threadsPerBlock = 128;
const int blocksPerGrid = 512;

__global__ void produttoria(Lock mylock, float*V, int N, float *ris){

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;	//indice complessivo del thread
  int cacheIndex = threadIdx.x;   				//indice nelle somme parziali del blocco.
  unsigned int Nthreads = blockDim.x * gridDim.x;	 	//Numero totale di threads istanziati dalla chiamata del kernel


//Fase 1; concentro il vettore in uno che ha tanti elementi quanti sono i blocchi
  __shared__ float Prodotto_parziale[threadsPerBlock]; //il blocco si alloca la sua bella memoria shared il valore deve essere una costante!
  Prodotto_parziale[cacheIndex]=V[tid];                 //ogni thread del blocco la riempie con un valore iniziale

  unsigned int k=tid+Nthreads;	
  while(k<N){// se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
	Prodotto_parziale[cacheIndex]=Prodotto_parziale[cacheIndex]*V[k];
	k+= Nthreads;	      
  } 
__syncthreads();//Sincronizzo per assicurarmi che ogni thread del blocco abbia caricato gli elementi.


//Fase 2; calcolo la produttoria di ogni blocco con metodo telescopico
 //si puo' quindi passare a valutare la somma parziale dei risultati dei vari threads di un blocco
  int i=blockDim.x/2;
// divido i thread del blocco in 2 meta'

  while(i!=0){
    if (cacheIndex < i) // i thread che sono nella prima meta' li produco con il corrispondente nella seconda meta'. solo una meta' dei thread lavoro effettivamente ma comunque partono tutti insieme! e' il parallelismo!
      //questa somma combina i valori della meta' superiore del blocco di threads con quelli della 
      //meta' inferiore e li associa a indici della meta' inferiore
      Prodotto_parziale[cacheIndex]*=Prodotto_parziale[cacheIndex+i];
    //e' necessario attendere che tutti i threads abbiano effettuato la somma
    __syncthreads(); // visto che solo meta' dei thread lavoro e necessario che si sincronizzano
    //tutti gli elementi rilevanti hanno indici da zero a meta' del blocco
    //nella seconda iterazione si sommano gli elementi del secondo quarto con quelli del primo
    //quarto del blocco, e cosi' via
    i/=2; // dimezzo ancora il blocco
    //poiche' i e' una variabile intera, nel caso 1/2 = 0.5 viene restituito 0
  };

//A questo punto nell'elemento Prodotto_parziale[0] di ogni blocco si trova una somma parziale


//Fase 3; ogni blocco accumula il valore in una variabile nella global (il primo che arriva agisce)
 if(cacheIndex==0){
	 mylock.lock() ;
	 *ris=*ris*Prodotto_parziale[threadIdx.x];
 	mylock.unlock() ;
 }

}


int main(){

//Definisco i puntatori alle variabli host
	float *A_host; 
//Definisco i puntatori alle variabli device
	float *A_dev; float *value_dev;

//alloco le variabili host
  	int N=1000000;  
	A_host = new float[N] ();
	float value_host;

//Alloco le variabili device
  hipMalloc( (void **)&A_dev, N*sizeof(float)  );
  hipMalloc( (void **)&value_dev, sizeof(float)  );

//riempio le variabili Device
        srand (time(NULL));	/* initialize random seed: */			 
        for(int i=0; i<N; i++)A_host[i]= ((float) rand() /RAND_MAX)*(0.01)+0.995;//genero tanti random molto vicini a 1
	hipMemcpy( A_dev, A_host, N*sizeof(float), hipMemcpyHostToDevice);

	value_host =1;
	hipMemcpy( value_dev , &value_host, sizeof(float), hipMemcpyHostToDevice);

        for(int i=0; i<N; i++)value_host*=A_host[i];
	cout<<"produttoria secondo la cpu :"<< value_host<<endl;


//Setup della configurazione di esecuzione

	dim3 dimBlock(threadsPerBlock,1);
	dim3 dimGrid(blocksPerGrid,1);

//Il lock lo passo come argomento del kernel
	Lock lock;

//Riempio le variabili device
	produttoria<<<dimGrid,dimBlock>>>( lock,A_dev, N, value_dev);
//riempio le variabili host
	hipMemcpy( &value_host, value_dev, sizeof(float), hipMemcpyDeviceToHost);
	cout<<"produttoria secondo la gpu :"<< value_host<<endl;


  return 0;
}


