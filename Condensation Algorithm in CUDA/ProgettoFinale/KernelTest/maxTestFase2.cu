#include "hip/hip_runtime.h"

 #include <iostream>
using namespace std;
#include <cstdlib>

#include "hip/hip_runtime.h"
#include "lock.h"


#define THREAD_PER_BLOCK 2
#define BLOCK_PER_GRID 2

#include "max.cuh"

int main( void ){


  unsigned int threadsPerBlock= THREAD_PER_BLOCK;
  unsigned int blocksPerGrid = BLOCK_PER_GRID;
  unsigned int Nthread=THREAD_PER_BLOCK*BLOCK_PER_GRID;

 //Definisco i puntatori alle variabli host
	float *A_host;
	float *A_test_host;
	int   *Index_test_host;
//Definisco i puntatori alle variabli device
	float *A_dev;
	float *A_test_dev;
	int   *Index_test_dev;



//alloco le variabili host
  	int N=7;  
	A_host = new float[N] ();
	A_test_host = new float[Nthread] ();
	Index_test_host = new int[Nthread] ();

//Alloco le variabili device
  	hipMalloc( (void **)&A_dev, N*sizeof(float)  );
 
  	hipMalloc( (void **)&A_test_dev, Nthread*sizeof(float)  );
  	hipMalloc( (void **)&Index_test_dev, Nthread*sizeof(int)  );





//Riempio le variabili host
  for( int i=0; i<N; i++){
    A_host[i] = -N+2*N*i;
    cout<<"A_host[ "<<i<<" ] = "<<A_host[i]<<endl;
  };

//Riempio le variabili Device
 hipMemcpy( A_dev,  A_host, N*sizeof(float), hipMemcpyHostToDevice);


//Testo fase 1, compressione del vettorone ad tot blocchi di tot thread
cout<<"FASE 1"<<endl;

  MaxValueFase1<<<blocksPerGrid,threadsPerBlock>>>(A_dev,N,
							A_test_dev,Index_test_dev);

//Pull back dei risultati
  hipMemcpy( A_test_host,  A_test_dev, Nthread*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy( Index_test_host,  Index_test_dev, Nthread*sizeof(int), hipMemcpyDeviceToHost);

  for( int i=0; i<Nthread; i++){
 
    cout<<"tread[ "<<i<<"] , idx = "<< Index_test_host[i] <<" val = "<<A_test_host[i] <<endl;
  }

//Testo fase 2, compressione del vettorone ad tot blocchi di tot thread
cout<<"FASE 2"<<endl;

  MaxValueFase2<<<blocksPerGrid,threadsPerBlock>>>(A_dev,N,
							A_test_dev,Index_test_dev);

//Pull back dei risultati
  hipMemcpy( A_test_host,  A_test_dev, Nthread*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy( Index_test_host,  Index_test_dev, Nthread*sizeof(int), hipMemcpyDeviceToHost);

  for( int i=0; i<Nthread; i++){
 
    cout<<A_test_host[i]<<" "<< Index_test_host[i] <<endl;
  }


  return 0;
}




