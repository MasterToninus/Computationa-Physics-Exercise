#include "hip/hip_runtime.h"
/*//---------------------------------------------------------------------------------------------------------
| faccio eseguire il calcolo del prodotto tra matrici
| 
*///---------------------------------------------------------------------------------------------------------

#define Tile_size=32


//kernel che intende che io ho allocato su gpu le 2 matrci A, B da produrre e la matrice C in cui mettere il risultato
// quindi si attende che numAColumns = numBRows; numCRows = numARows ; numCColumns = numAColumns
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns) 
{
    __shared__ float A_shared[Tile_size][Tile_size];   // Tile size of 32x32 
    __shared__ float B_shared[Tile_size][Tile_size];

    unsigned int Thread_i = threadIdx.y;
    unsigned int Thread_j = threadIdx.x;	

    unsigned int Tile_i = blockIdx.y*Tile_size;	     //indici del primo elemento della tile (
    unsigned int Tile_j = blockIdx.x*Tile_size;  	


    while(Tile_j<numCColumns)
    {
	    while(Tile_i<numCRows){
		//... conto del tile	//ogni thread calcola 1 o piu' elementi della matrice c
		    float Cvalue = 0.0; 		
		    A_shared[Thread_i][Thread_j] = 0.0;
		    B_shared[Thread_i][Thread_j] = 0.0;



		
		// riempio la memoria shared necessaria per il calcolo del tile
	//ricordare, per ogni tile della matrice C servono due liste di tile: una orizzontale nella matrice A e una verticale nella matrice B
		    for (int k = 0; k < (((numAColumns - 1)/ Tile_size) + 1); k++)
    		    {	
			//indice dell elemento di matrice C corrispondente al dato thread
			unsigned int Mat_i = Tile_i + Thread_i , Mat_j = Tile_j + Thread_j ;	
			
		        if ( (Mat_i < numARows) && (Thread_j + (k*32)) < numAColumns)
		        {
		            A_shared[Thread_i][Thread_j] = A[(Mat_i*numAColumns) + threadIdx.x + (k*32)];
		        }
		        else
		        {
		            A_shared[Thread_i][Thread_j] = 0.0;
		        }            
		        if ( (Mat_j < numBColumns) && (Thread_i + k*32) < numBRows)
		        {
		            B_shared[Thread_i][Thread_j] = B[(threadIdx.y + k*32)*numBColumns + Mat_j];
		        }
		        else
		        {
		            B_shared[Thread_i][Thread_j]= 0.0;
		        }            
		        __syncthreads();

		        for (int j = 0; j < Tile_size; ++j)
		        {
		            Cvalue += A_shared[Thread_i][j] * B_shared[j][Thread_j];
		        }
 
			if (Mat_i < numCRows && Mat_j < numCColumns)
    			{	
		            C[Mat_i*numCColumns + Mat_j] = Cvalue;
    			}


   		  }
		Tile_i+=Tile_size*gridDim.y;
	    }	
	Tile_j+=Tile_size*gridDim.x;
    }	


}



/*
// SCOPIAZZATO
*/



#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// This code assumes that your device support block size of 1024
#define MAX_RANGE 9999

#define funcCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)




void matMultiplyOnHost(float * A, float * B, float * C, int numARows,
                        int numAColumns, int numBRows, int numBColumns,
                        int numCRows, int numCColumns)
{
    for (int i=0; i < numARows; i ++)
    {
        for (int j = 0; j < numAColumns; j++)
        {
            C[i*numCColumns + j ] = 0.0;
            for (int k = 0; k < numCColumns; k++)
            {
                C[i*numCColumns + j ] += A[i*numAColumns + k] * B [k*numBColumns + j];
            }
        }
    }
    return;
}







int main() {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;

    // Please adjust rows and columns according to you need.
    int numARows = 12; // number of rows in the matrix A
    int numAColumns = 15; // number of columns in the matrix A
    int numBRows = 15; // number of rows in the matrix B
    int numBColumns = 3; // number of columns in the matrix B

    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)
    {
        hostA[i] = (rand() % MAX_RANGE) / 2.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i] = (rand() % MAX_RANGE) / 2.0;
    }

    // Setting numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);    
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);    

    // Allocating GPU memory
    funcCheck(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
    funcCheck(hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns));
    funcCheck(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));

    // Copy memory to the GPU 
    funcCheck(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions 
    dim3 dimBlock(32, 32, 1);    
    dim3 dimGrid((numCColumns/32) + 1, (numCRows/32) + 1, 1);

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);    

    hipError_t err1 = hipPeekAtLastError();
    hipDeviceSynchronize();
    printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    // Copy the results in GPU memory back to the CPU    
    funcCheck(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

    matMultiplyOnHost(hostA, hostB, hostComputedC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    for (int i=0; i < numCColumns*numCRows; i++)
    {
        if (hostComputedC[i]  != hostC[i] )
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns, i % numCColumns, hostComputedC[i], hostC[i]);
            break;
        }
    }
    // Free the GPU memory
    funcCheck(hipFree(deviceA));
    funcCheck(hipFree(deviceB));        
    funcCheck(hipFree(deviceC));    

    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}

