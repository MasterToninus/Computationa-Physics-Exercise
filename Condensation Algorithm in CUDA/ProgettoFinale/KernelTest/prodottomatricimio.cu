
#include <hip/hip_runtime.h>
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns) 
{
    __shared__ float sA[32][32];   // Tile size of 32x32 
    __shared__ float sB[32][32];

    int Row = blockDim.y*blockIdx.y + threadIdx.y;
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ 32) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*32)) < numAColumns)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*32)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }            
        if ( Col < numBColumns && (threadIdx.y + k*32) < numBRows)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*32)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }            
        __syncthreads();

        for (int j = 0; j < 32; ++j)
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}

