#include "hip/hip_runtime.h"
/*//---------------------------------------------------------------------------------------------------------
|
|  Test dell'ultima versione del kernel, usa una shared  larga come il warp e una copia texture dell'intera matrice.
|  questa ultima versione e' pensata per essere usata con blocchi 2D 
|
*///---------------------------------------------------------------------------------------------------------
//OGNI thread alloca 5 unsigned int e 2 float ---> 7*4= 28 byte



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/


// Numero di thread prudente
#define DIM_GRID 128
#define DIM_BLOCK 128
#define THREAD_PER_BLOCK 128



/*     DIMENSIONALIZZAZIONE DEI THREAD    */
// Numero di thread "Limite"
#define imin(a,b) (a<b?a:b)

//trucco per ottimizzare il numero di blocchi
//const int blocksPerGrid = imin(32,(N+threadsPerBlock-1)/threadsPerBlock);

#define MAX_DIM_BLOCK 512
#define MAX_DIM_GRID 524288 //in realtà il max numero di thrad è un po' di piu', questa è la potenza di 2 più vicina
#define DIM_WARP 32


#include "condensation_tentative.cuh"




void print(float *A, int ROW, int COL){
		for(int i=0 ; i<ROW ; i++){
			for(int j=0 ; j<COL ; j++){
				//printf(" %.1f ",A[i*COL+j]);
				cout<< A[i*COL+j] << " ";
			}
			cout<<endl;
		}
		cout<<endl;  
}







int main(void){

	int ROW=10000;
	int COL=ROW;
	int N = COL*ROW;
	int Nb= (ROW-1)*(COL-1);
	srand (time(NULL));  

   //Punto La memoria
	float * A_host; float *B_host;
	float * A_dev; float *B_dev;

   //Alloco la matrice
	A_host = new float[N] ();
	B_host = new float[Nb] ();


   //Alloco le matrice device (riservo la memoria richiesta)
	unsigned int ByteSize = N*sizeof(float);
	hipMalloc( (void **)&A_dev, ByteSize  );
	ByteSize = Nb*sizeof(float);
	hipMalloc( (void **)&B_dev, ByteSize  );


   //Riempio la Matrice Host
	//float min = -10, max = 10 , delta = max-min; 
	for(int i=0 ; i<ROW ; i++)for(int j=0 ; j<COL ; j++)A_host[i*COL+j] =  (rand() % 10 );
        A_host[N-1]=1;

   //Riempio La Matrice Device
	hipMemcpy( A_dev, A_host, N*sizeof(float), hipMemcpyHostToDevice);

   //Check Matrice Creata
	//	print(A_host,ROW,COL);



	//Creo degli eventi per la statistica
	hipEvent_t T1,T2;
	hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;




cout<<"CONDENSAZIONE VERSIONE 6 (1 sola  texture grande)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------

        dim3 block3(DIM_BLOCK );  // 512 x 1 x 1    

        unsigned int block_per_row = DIM_GRID;
        unsigned int block_per_col = 1;// imin(COL, MAX_DIM_GRID / block_per_row);
        dim3 grid3( block_per_row, block_per_col );         // 512 x 1 x 1

        cout<<"blocchi : "<<DIM_BLOCK<<" x 1 x 1 "<<endl;
        cout<<"griglia : "<< block_per_row <<" x "<< block_per_col << " x 1"<<endl;



   //Kernel
	stepCondensationSimple_v6<<<grid3,block3>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);


// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef2 );
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//		print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;



cout<<"CONDENSAZIONE VERSIONE 6 (1 sola  texture grande) versione ottimizzata"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------

	unsigned int thread_per_row = DIM_WARP;//imin(COL,MAX_DIM_BLOCK);
        dim3 block4(thread_per_row );  // 512 x 1 x 1    

        block_per_row = ROW/THREAD_PER_BLOCK + 1;
        block_per_col =  imin(COL, MAX_DIM_GRID / block_per_row);
        dim3 grid4( block_per_row, block_per_col );         // 512 x 1 x 1

        cout<<"blocchi : "<<thread_per_row<<" x 1 x 1 "<<endl;
        cout<<"griglia : "<< block_per_row <<" x "<< block_per_col << " x 1"<<endl;



   //Kernel
	stepCondensationSimple_v6<<<grid4,block4>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);


// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef2 );
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//		print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;



cout<<"CONDENSAZIONE VERSIONE 7 (1 sola  texture grande e la shared) versione ottimizzata"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------



	thread_per_row = DIM_WARP; 
        block_per_row = ROW/thread_per_row + 1;
	unsigned int thread_per_col = MAX_DIM_BLOCK/DIM_WARP;

        block_per_col =  1;//imin(COL, MAX_DIM_GRID / block_per_row);

        dim3 block0(thread_per_row, thread_per_col );  
        dim3 grid0( block_per_row, block_per_col );         // 512 x 1 x 1

        cout<<"blocchi : "<< thread_per_row <<" x "<< thread_per_col <<"  x 1 "<<endl;
        cout<<"griglia : "<< block_per_row <<" x "<< block_per_col << " x 1"<<endl;



   //Kernel
	stepCondensationSimple_v7<<<grid0,block0>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);


// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef2 );
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//		print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;





   hipFree( A_dev);
   hipFree( B_dev);



  return 0;
}

