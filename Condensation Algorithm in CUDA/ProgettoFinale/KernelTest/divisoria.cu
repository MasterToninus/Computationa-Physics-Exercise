
 #include <iostream>
using namespace std;
#include <cstdlib>


#define THREAD_PER_BLOCK 128
#define BLOCK_PER_GRID 128

#include "divisoria.cuh"

int main( void ){


  //Definisco i puntatori alle variabli host
	float *A_host;

//Definisco i puntatori alle variabli device
	float *A_dev;



//alloco le variabili host
  	int N=15;  
	A_host = new float[N] ();

//Alloco le variabili device
  	hipMalloc( (void **)&A_dev, N*sizeof(float)  );
 
//Riempio le variabili host
  for( int i=0; i<N; i++)A_host[i] = drand48()*10;
  cout<<endl;for( int i=0; i<N; i++) cout<<A_host[i]<<endl;

//Riempio le variabili Device
 hipMemcpy( A_dev,  A_host, N*sizeof(float), hipMemcpyHostToDevice);

//Testo divisoria Gpu
	divisoria_Cuda_v1<<<N,1>>>(A_host,10.0,N);

//Pull back dei risultati
  hipMemcpy( A_host,  A_dev, N*sizeof(float), hipMemcpyDeviceToHost);

//Testo divisoria Cpu
	divisoria_Cpu_v1(A_host,10.0,N);
  cout<<endl;for( int i=0; i<N; i++) cout<<A_host[i]<<endl;





  cout<<endl;for( int i=0; i<N; i++) cout<<A_host[i]<<endl;


  return 0;
}





