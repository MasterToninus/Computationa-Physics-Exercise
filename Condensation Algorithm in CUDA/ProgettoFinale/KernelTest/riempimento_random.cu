/*//---------------------------------------------------------------------------------------------------------
| faccio eseguire un semplice riempimento di vettore alla scheda, riempie con l'indice del thread
| stare attenti pero' al limite di memoria allocabile!
| 
*///---------------------------------------------------------------------------------------------------------






#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>


// IMPLEMENTO I RANDOOM!
#include <hip/hip_runtime_api.h>	//tutto cio per richiamare il curand
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <ctime>		//per inizializzare il seed con time

__global__ void initRandom(hiprandState *  State, const unsigned int seed){

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, tid, 0, &State[tid]);

}



__global__ void riempimento_random(float*V, int N, hiprandState *  State, float Min, float Max){

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int Nthreads = blockDim.x * gridDim.x;	 //Numero totale di threads istanziati dalla chiamata del kernel

  unsigned int k=tid;	
 
  hiprandState localState = State[tid];

  while(k<N){
	V[k]= Min + hiprand_uniform(&localState) *(Max-Min);
	k+= Nthreads;	      // se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
  } 

  State[tid]=localState;


}




int main(){

//Definisco i puntatori alle variabli host
	float *A_host; 
//Definisco i puntatori alle variabli device
	float *A_dev;

//alloco le variabili host
  	int N=100000000;  
	A_host = new float[N] ();

//Alloco le variabili device
  hipMalloc( (void **)&A_dev, N*sizeof(float)  );

//Setup della configurazione di esecuzione
/*	cudaDeviceProp prop; //cudaDeviceProp e' una struttura di dati, prop e' il nome della variabile di questo tipo
	cudaGetDeviceProperties( &prop, 0); //salva in prop le caratteristiche della prima scheda
*/
	unsigned int Block_Width=128;//prop.maxThreadsDim[0]/2;
	unsigned int Grid_Width=512;//prop.maxGridSize[0]/2;
/*
	dim3 dimBlock(Block_Width,1);
	dim3 dimGrid(Grid_Width,1);
*/
	unsigned int nThreads=Block_Width*Grid_Width;

//Inizializzo i seed per il random ogni thread avra' il suo seed
	hiprandState *dev_State ;
	hipMalloc( (void **)&dev_State, nThreads*sizeof(hiprandState) );
	int seed =time( 0 );
//Riempio le variabili device

	initRandom<<<Grid_Width,Block_Width>>>(dev_State,seed);
	//riempimento_random<<<dimGrid,dimBlock>>>(A_dev,N,dev_State,seed,0.5,2.);
	riempimento_random<<<Grid_Width,Block_Width>>>(A_dev,N,dev_State,0.5,2.);
	
//riempio le variabili host
	hipMemcpy( A_host, A_dev, N*sizeof(float), hipMemcpyDeviceToHost);

//controllo!
	for(int i=0; i<N; i++)printf("A[%d] = %f \n",i,A_host[i]);

  //Disalloco la memoria riservata sulla scheda
	hipFree( dev_State);
	hipFree( A_dev);


  return 0;
}


