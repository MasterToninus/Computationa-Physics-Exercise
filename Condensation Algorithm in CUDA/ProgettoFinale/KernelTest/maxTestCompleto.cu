#include "hip/hip_runtime.h"

 #include <iostream>
using namespace std;
#include <cstdlib>

#include "hip/hip_runtime.h"
#include "lock.h"


#define THREAD_PER_BLOCK 128
#define BLOCK_PER_GRID 64

#include "max.cuh"

int main( void ){


  unsigned int threadsPerBlock= THREAD_PER_BLOCK;
  unsigned int blocksPerGrid = BLOCK_PER_GRID;
  unsigned int Nthread=THREAD_PER_BLOCK*BLOCK_PER_GRID;

 //Definisco i puntatori alle variabli host
	float *A_host;
//Definisco i puntatori alle variabli device
	float *A_dev;
	float *max_Value_dev;
	int   *max_Index_dev;



//alloco le variabili host
  	int N=10000;  
	A_host = new float[N] ();
	float max_Value_host;
	int   max_Index_host;

//Alloco le variabili device
  	hipMalloc( (void **)&A_dev, N*sizeof(float)  );

 	hipMalloc( (void **)&max_Value_dev, sizeof(float)  );
  	hipMalloc( (void **)&max_Index_dev, sizeof(int)  );
 
//Riempio le variabili host
  for( int i=0; i<N; i++) A_host[i] = drand48();
  A_host[17]=-100.9;
  for( int i=0; i<N; i++) cout<<"A_host[ "<<i<<" ] = "<<A_host[i]<<endl;

    max_Value_host = A_host[0];
    max_Index_host=0;


//Riempio le variabili Device
  hipMemcpy( A_dev,  A_host, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy( max_Value_dev,  &max_Value_host, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy( max_Index_dev,  &max_Index_host, sizeof(int), hipMemcpyHostToDevice);

//KERNEL!

	Lock lock;
  MaxValueCompleto<<<blocksPerGrid,threadsPerBlock>>>(lock,A_dev,N,
							max_Value_dev,max_Index_dev);

//Pull back dei risultati
  hipMemcpy( &max_Value_host,  max_Value_dev, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy( &max_Index_host,  max_Index_dev, sizeof(int), hipMemcpyDeviceToHost);



cout<<"max ="<<max_Value_host<<" idx = "<< max_Index_host <<endl;
  return 0;
}




