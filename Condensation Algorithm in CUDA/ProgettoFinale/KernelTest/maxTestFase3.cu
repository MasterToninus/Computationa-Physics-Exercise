#include "hip/hip_runtime.h"

 #include <iostream>
using namespace std;
#include <cstdlib>

#include "hip/hip_runtime.h"
#include "lock.h"


#define THREAD_PER_BLOCK 128
#define BLOCK_PER_GRID 64

#include "max.cuh"

int main( void ){


  unsigned int threadsPerBlock= THREAD_PER_BLOCK;
  unsigned int blocksPerGrid = BLOCK_PER_GRID;
  unsigned int Nthread=THREAD_PER_BLOCK*BLOCK_PER_GRID;

 //Definisco i puntatori alle variabli host
	float *A_host;
	float *A_test_host;
	int   *Index_test_host;
//Definisco i puntatori alle variabli device
	float *A_dev;
	float *A_test_dev;
	int   *Index_test_dev;



//alloco le variabili host
  	int N=10000;  
	A_host = new float[N] ();
	A_test_host = new float[Nthread] ();
	Index_test_host = new int[Nthread] ();

//Alloco le variabili device
  	hipMalloc( (void **)&A_dev, N*sizeof(float)  );
 
  	hipMalloc( (void **)&A_test_dev, Nthread*sizeof(float)  );
  	hipMalloc( (void **)&Index_test_dev, Nthread*sizeof(int)  );





//Riempio le variabili host
  for( int i=0; i<N; i++) A_host[i] = 0;
  A_host[17]=-3.5;
  for( int i=0; i<N; i++) cout<<"A_host[ "<<i<<" ] = "<<A_host[i]<<endl;


//Riempio le variabili Device
 hipMemcpy( A_dev,  A_host, N*sizeof(float), hipMemcpyHostToDevice);


//Testo fase 1, compressione del vettorone ad tot blocchi di tot thread
cout<<"FASE 1"<<endl;

  MaxValueFase1<<<blocksPerGrid,threadsPerBlock>>>(A_dev,N,
							A_test_dev,Index_test_dev);

//Pull back dei risultati
  hipMemcpy( A_test_host,  A_test_dev, Nthread*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy( Index_test_host,  Index_test_dev, Nthread*sizeof(int), hipMemcpyDeviceToHost);

  for( int i=0; i<Nthread; i++){
 
    cout<<"tread[ "<<i<<"] , idx = "<< Index_test_host[i] <<" val = "<<A_test_host[i] <<endl;
  }

//Testo fase 2, compressione del vettorone ad tot blocchi di tot thread
cout<<"FASE 2"<<endl;

  MaxValueFase2<<<blocksPerGrid,threadsPerBlock>>>(A_dev,N,
							A_test_dev,Index_test_dev);

//Pull back dei risultati
  hipMemcpy( A_test_host,  A_test_dev, Nthread*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy( Index_test_host,  Index_test_dev, Nthread*sizeof(int), hipMemcpyDeviceToHost);

  for( int i=0; i<Nthread; i++){
 
    cout<<"tread[ "<<i<<"] , idx = "<< Index_test_host[i] <<" val = "<<A_test_host[i] <<endl;
  }


//Testo fase 3
cout<<"FASE 3"<<endl;

	Lock lock;
  MaxValueFase3<<<blocksPerGrid,threadsPerBlock>>>(lock,A_dev,N,
							A_test_dev,Index_test_dev);

//Pull back dei risultati
  hipMemcpy( A_test_host,  A_test_dev, Nthread*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy( Index_test_host,  Index_test_dev, Nthread*sizeof(int), hipMemcpyDeviceToHost);

  for( int i=0; i<Nthread; i++){
 
    cout<<"tread[ "<<i<<"] , idx = "<< Index_test_host[i] <<" val = "<<A_test_host[i] <<endl;
  }

//Testo fase Conclusione
cout<<"Conclusione"<<endl;
cout<<A_test_host[0]<<" "<< Index_test_host[0] <<endl;
  return 0;
}




