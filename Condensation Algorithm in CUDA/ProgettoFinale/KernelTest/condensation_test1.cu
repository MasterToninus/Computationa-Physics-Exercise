#include "hip/hip_runtime.h"
/*//---------------------------------------------------------------------------------------------------------
|
|  Test di Kernel simili che utilizzano memoria in modo diverso ( mostrare il diverso tempo di esecuzione)
|
*///---------------------------------------------------------------------------------------------------------
//OGNI thread alloca 5 unsigned int e 2 float ---> 7*4= 28 byte



#include <iostream>
#include <stdio.h>
using namespace std;
#include <cstdlib>
#include <stdlib.h>     /* srand, rand */ //http://www.cplusplus.com/reference/cstdlib/rand/
#include <fstream>	//http://www.cplusplus.com/doc/tutorial/files/



#define DIM_GRID 128
#define DIM_BLOCK 128
#define THREAD_PER_BLOCK 128
#include "condensation_tentative.cuh"

//kernel che copia una colonna della matrice in un segmento lineare di memoria
/*
input: 
	puntatore destinatatio
	puntatore matricepartenza
	indice della colonna da copiare
	numero di elementi
*/

__global__ void copy_Col( float *a, float *A,int j1, int numAColumns, int numARow){

  int tid = blockIdx.x*blockDim.x + threadIdx.x; //identità del thread (al solito metto in fila ogni thread a partire dal primo blocco)
  int Nthreads = blockDim.x * gridDim.x;	 //Numero totale di threads istanziati dalla chiamata del kernel

  while(tid<numARow){
	a[tid]=A[tid*numAColumns + j1]; //qui copia
	tid+= Nthreads;	      // se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
	}
  
}

void print(float *A, int ROW, int COL){
		for(int i=0 ; i<ROW ; i++){
			for(int j=0 ; j<COL ; j++){
				//printf(" %.1f ",A[i*COL+j]);
				cout<< A[i*COL+j] << " ";
			}
			cout<<endl;
		}
		cout<<endl;  
}







int main(void){

	int ROW=10000;
	int COL=ROW;
	int N = COL*ROW;
	int Nb= (ROW-1)*(COL-1);
	srand (time(NULL));  

   //Punto La memoria
	float * A_host; float *B_host;
	float * A_dev; float *B_dev;

   //Alloco la matrice
	A_host = new float[N] ();
	B_host = new float[Nb] ();


   //Alloco le matrice device (riservo la memoria richiesta)
	unsigned int ByteSize = N*sizeof(float);
	hipMalloc( (void **)&A_dev, ByteSize  );
	ByteSize = Nb*sizeof(float);
	hipMalloc( (void **)&B_dev, ByteSize  );


   //Riempio la Matrice Host
	//float min = -10, max = 10 , delta = max-min; 
	for(int i=0 ; i<ROW ; i++)for(int j=0 ; j<COL ; j++)A_host[i*COL+j] =  (rand() % 10 );
        A_host[N-1]=1;

   //Riempio La Matrice Device
	hipMemcpy( A_dev, A_host, N*sizeof(float), hipMemcpyHostToDevice);

   //Check Matrice Creata
	//	print(A_host,ROW,COL);



	//Creo degli eventi per la statistica
	hipEvent_t T1,T2;
	hipEventCreate(&T1);
	hipEventCreate(&T2);
	float diff_time;




cout<<"CONDENSAZIONE VERSIONE 1"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

   //Kernel
	stepCondensationSimple_v1<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;

cout<<"CONDENSAZIONE VERSIONE 2"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

   //Kernel
	stepCondensationSimple_v2<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;

cout<<"CONDENSAZIONE VERSIONE 3"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);
   //Kernel
	stepCondensationSimple_v3<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1); 
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;

cout<<"CONDENSAZIONE VERSIONE 4 (1 texture)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	float *pivot_Column = 0;
	unsigned int numBytes = ROW*sizeof(float);
	//Alloco la memoria lineare sulla gpu dove caricare la colonna di pivot
	hipMalloc((void**)&pivot_Column, numBytes);

	//Chiamo kernel che mette la colonna in memoria lineare
	copy_Col<<<DIM_GRID,DIM_BLOCK>>>(pivot_Column, A_dev ,COL-1, COL, ROW);

	//Bindo a texture la memoria lineare
	hipBindTexture(NULL, texRef, pivot_Column);
// --------- 


   //Kernel
	stepCondensationSimple_v4<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);

// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef );
	hipFree (pivot_Column);
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;



cout<<"CONDENSAZIONE VERSIONE 5 (2 texture)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	// *pivot_Column = 0;
	 numBytes = ROW*sizeof(float);
	//Alloco la memoria lineare sulla gpu dove caricare la colonna di pivot
	hipMalloc((void**)&pivot_Column, numBytes);

	//Chiamo kernel che mette la colonna in memoria lineare
	copy_Col<<<DIM_GRID,DIM_BLOCK>>>(pivot_Column, A_dev ,COL-1, COL, ROW);

	//Bindo a texture la memoria lineare
	hipBindTexture(NULL, texRef, pivot_Column);
// --------- 
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------

   //Kernel
	stepCondensationSimple_v5<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);



// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef );
	hipUnbindTexture ( texRef2 );
	hipFree (pivot_Column);
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;




cout<<"CONDENSAZIONE VERSIONE 6 (1 sola  texture grande)"<<endl; 
	//registro il primo tempo
	hipEventRecord(T1, 0);

// --------- Preparo Texture
	//Bindo a texture tutta la matrice in modo che e' bindato anche il vettore che mi interessa
	 hipBindTexture(NULL, texRef2, A_dev);
// ---------

   //Kernel
	stepCondensationSimple_v6<<<DIM_GRID,DIM_BLOCK>>>(B_dev,A_dev, COL-1,ROW-1);		
   //PullBack del risultato
	hipMemcpy( B_host, B_dev, Nb*sizeof(float), hipMemcpyDeviceToHost);


// --------- Preparo Texture
	//unbind della texture
	hipUnbindTexture ( texRef2 );
// --------- 

	//registro il secondo evento alla fine del kernel
	hipEventRecord(T2, 0);
	hipEventSynchronize(T2);

   //check
	//	print(B_host,ROW-1,COL-1);
	hipEventElapsedTime(&diff_time, T1, T2);
	cout<<"Tempo esecuzione = "<<diff_time<<" ms"<<endl;



   hipFree( A_dev);
   hipFree( B_dev);



  return 0;
}

