#include "hip/hip_runtime.h"
/*//---------------------------------------------------------------------------------------------------------
| kernel ausiliari
| 
*///---------------------------------------------------------------------------------------------------------


__global__ void swapRow(float * A, int i1, int i2, int numAColumns) 
{
// OGNI thread copia un elemento di riga nel suo registo ed esegue lo swap
   unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
   unsigned int Nthreads = blockDim.x * gridDim.x;	
   float temp;

  while(tid<numAColumns){
	temp= A[i1*numAColumns + tid];
	A[i1*numAColumns + tid] = A[i2*numAColumns + tid];
	A[i2*numAColumns + tid] = temp;
	tid+= Nthreads;	      // se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
  }

}

#ifndef MAX_FLOAT(a,b)
#define MAX_FLOAT(a,b) (a>b?a:b)


//fare attenzione! non lanciare piu' thread del necessario! Nthread sempre minore del numero di componenti!
// kernel si aspetta: 	Nthread totali minore del numero di elementi
//			Nthread per blocco potenza di 2

__global__ void MaxValueinRow(float * A, int i1, int numAColumns) 
{
   unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;	//indice complessivo del thread
   int cacheIndex = threadIdx.x;   				//indice nelle somme parziali del blocco.
   unsigned int Nthreads = blockDim.x * gridDim.x;	 	//Numero totale di threads istanziati dalla chiamata del kernel

   //fase zero, ogni thread si copia nella shared: 
   __shared__ float A_shared[blockDim.x]; 		//il valore assoluto del vettore corrispondente al tid
   __shared__ unsigned int Index_shared[blockDim.x];	// l'indice come elemento di matrice dell'elemento salvato nella shared
//   if(tid<numAColumns) //questo if e' superfluo! questo kernel presume di lanciare sempre un numero di thread minore o uguale al numero di elementi!
   float temp=A[tid];
   if(temp<0)temp=-temp;  
   A_shared[cacheIndex]=temp;
   Index_shared[cacheIndex]=tid;

   //Fase 1; concentro il vettore in uno che ha tanti elementi quanti sono i blocchi ogniuno contenente una potenza di 2 di elementi (quindi telescopicizzabile)http://www.cuvilib.com/Reduction.pdf

   unsigned int dummyid=tid+Nthreads;

  while(dummyid<numAColumns){
		temp=A[dummyid];
   		if(temp<0)temp=-temp;
		if(A_shared[cacheIndex]<temp){
			A_shared[cacheIndex]=temp;
			Index_shared[cacheIndex]=dummyid;
		}

	dummyid+= Nthreads;	      // se il vettore ha più elementi del numero di thread chiamati faccio si che ogni thread sommi più di un elemento
  }
  __syncthreads();//Sincronizzo per assicurarmi che ogni thread del blocco abbia caricato gli elementi.


   //Fase 2, riduco in modo telescopico il vettore
   dummyid=blockDim.x/2;
  while(dummyid!=0){
    if (cacheIndex < dummyid)
      A_shared[cacheIndex]=MAX_FLOAT(A_shared[cacheIndex],A_shared[cacheIndex+dummyid]);
      __syncthreads(); 
       dummyid/=2;
  }

//A questo punto nell'elemento A_shared[0] di ogni blocco si trova un massimo parziale
   //Fase 3, stabilisco qual'e' il massimo di ogni blocco







}

